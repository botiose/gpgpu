#include <iostream>

#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "kernel.cuh"

#define GRID_SIZE 32

#define POOL_SIZE 3
#define ERO_KERNEL_WIDTH 6
#define ERO_KERNEL_HEIGHT 3
#define DIL_KERNEL_WIDTH 30
#define DIL_KERNEL_HEIGHT 15

unsigned int
divUp(const unsigned int& a, const unsigned int& b) {
  if (a % b != 0) {
    return a / b + 1;
  } else {
    return a / b;
  }
}

int
main(int argc, char** argv) {
  if (argc != 3) {
    std::cerr << "Usage: gpgpu <src-image.jpb> <dst-image.png>" << std::endl;
    exit(1);
  }

  int width, height, bpp;

  uint8_t* rgb = stbi_load(argv[1], &width, &height, &bpp, 3);

  int imageSize = sizeof(uint8_t) * (width * height);

  uint8_t* origDev;
  uint8_t* grayDev;
  hipMalloc((void**)&origDev, imageSize * 3);
  hipMalloc((void**)&grayDev, imageSize);
  hipMemcpy(origDev, rgb, imageSize * 3, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(GRID_SIZE, GRID_SIZE);
  dim3 numBlocks(divUp(width, GRID_SIZE), divUp(height, GRID_SIZE));

  // Compute the grayscale.
  grayscale<<<numBlocks, threadsPerBlock>>>(origDev, width, grayDev);

  stbi_image_free(rgb);

  hipDeviceSynchronize();

  hipFree(origDev);

  int pooledWidth = width / POOL_SIZE;
  int pooledHeight = height / POOL_SIZE;
  int pooledImageSize = sizeof(uint32_t) * (pooledWidth * pooledHeight);

  uint32_t* pooledXDev;
  uint32_t* pooledYDev;
  hipMalloc((void**)&pooledXDev, pooledImageSize);
  hipMalloc((void**)&pooledYDev, pooledImageSize);
  hipMemset(pooledXDev, 0, pooledImageSize);
  hipMemset(pooledYDev, 0, pooledImageSize);

  // Sobel and add to pool.
  sobelImage<<<numBlocks, threadsPerBlock>>>(
      grayDev, width, height, POOL_SIZE, pooledWidth, pooledXDev, pooledYDev);

  hipDeviceSynchronize();

  hipFree(grayDev);

  dim3 threadsPerBlockPool(GRID_SIZE, GRID_SIZE);
  dim3 numBlocksPool(divUp(pooledWidth, GRID_SIZE),
                     divUp(pooledHeight, GRID_SIZE));

  computeResponse<<<numBlocksPool, threadsPerBlockPool>>>(
      pooledYDev, POOL_SIZE, pooledWidth, pooledXDev);

  hipDeviceSynchronize();

  hipFree(pooledYDev);

  pooledImageSize = sizeof(uint8_t) * (pooledWidth * pooledHeight);

  uint8_t* erodedDev;
  hipMalloc((void**)&erodedDev, pooledImageSize);

  // Erode the image.
  erode<<<numBlocksPool, threadsPerBlockPool>>>(pooledXDev,
                                                pooledWidth,
                                                pooledHeight,
                                                ERO_KERNEL_WIDTH / 2,
                                                ERO_KERNEL_HEIGHT / 2,
                                                erodedDev);

  hipDeviceSynchronize();

  hipFree(pooledXDev);

  uint8_t* dilatatedDev;
  hipMalloc((void**)&dilatatedDev, pooledImageSize);

  // Dilatate the image.
  dilatate<<<numBlocksPool, threadsPerBlockPool>>>(erodedDev,
                                                   pooledWidth,
                                                   pooledHeight,
                                                   DIL_KERNEL_WIDTH / 2,
                                                   DIL_KERNEL_HEIGHT / 2,
                                                   dilatatedDev);

  hipDeviceSynchronize();

  hipFree(erodedDev);

  int eltCount = pooledWidth * pooledHeight;

  thrust::device_ptr<uint8_t> dev_ptr(dilatatedDev);
  int maxVal =
      thrust::reduce(dev_ptr, dev_ptr + eltCount, 0, thrust::maximum<int>());

  threshold<<<numBlocksPool, threadsPerBlockPool>>>(
      pooledWidth, maxVal, dilatatedDev);

  hipDeviceSynchronize();

  uint8_t* dilatatedHost = (uint8_t*)malloc(pooledImageSize);
  hipMemcpy(
      dilatatedHost, dilatatedDev, pooledImageSize, hipMemcpyDeviceToHost);

  hipFree(dilatatedDev);

  stbi_write_jpg(argv[2], pooledWidth, pooledHeight, 1, dilatatedHost, 100);

  free(dilatatedHost);
}
